#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "common.h"

#define VOLATILE(x) (*(volatile typeof(x) *)&(x))

__global__ void warmup_kernel(uintptr_t addr, size_t size)
{
    printf(">>>>>>> CUDA warmup! addr %p size %d\n", (void*)addr, (uint32_t)size);
}

__global__ void target_kernel(uintptr_t addr, size_t size)
{
    printf(">>>>>>> CUDA target waiting on addr %p size %d\n", (void*)addr, (uint32_t)size);
    while(VOLATILE(((uint8_t*)addr)[0]) == 0);
    printf(">>>>>>> CUDA target now addr %p is %d\n", (void*)addr, VOLATILE(((uint8_t*)addr)[0]));
}

__global__ void initiator_kernel(uintptr_t addr, size_t size)
{
    printf(">>>>>>> CUDA initiator send on addr %p size %d\n", (void*)addr, (uint32_t)size);
    for (int i = 0; i < (int)size; i++)
        ((uint8_t*)addr)[i] = 0x1;
}

extern "C" {

int launch_warmup_kernel(hipStream_t stream, uintptr_t addr, size_t size)
{
    hipError_t result = hipSuccess;

    /* Check no previous CUDA errors */
    result = hipGetLastError();
    if (result != hipSuccess) {
        fprintf(stderr, "[%s:%d] cuda failed with %s", __FILE__, __LINE__, hipGetErrorString(result));
        return -1;
    }

    warmup_kernel<<<1, 1, 0, stream>>>(addr, size);
    result = hipGetLastError();
    if (result != hipSuccess) {
        fprintf(stderr, "[%s:%d] cuda failed with %s", __FILE__, __LINE__, hipGetErrorString(result));
        return -1;
    }

    return 0;
}

int launch_target_wait_kernel(hipStream_t stream, uintptr_t addr, size_t size)
{
    hipError_t result = hipSuccess;

    /* Check no previous CUDA errors */
    result = hipGetLastError();
    if (result != hipSuccess) {
        fprintf(stderr, "[%s:%d] cuda failed with %s", __FILE__, __LINE__, hipGetErrorString(result));
        return -1;
    }

    target_kernel<<<1, 1, 0, stream>>>(addr, size);
    result = hipGetLastError();
    if (result != hipSuccess) {
        fprintf(stderr, "[%s:%d] cuda failed with %s", __FILE__, __LINE__, hipGetErrorString(result));
        return -1;
    }

    return 0;
}

int launch_initiator_send_kernel(hipStream_t stream, uintptr_t addr, size_t size)
{
    hipError_t result = hipSuccess;

    /* Check no previous CUDA errors */
    result = hipGetLastError();
    if (result != hipSuccess) {
        fprintf(stderr, "[%s:%d] cuda failed with %s", __FILE__, __LINE__, hipGetErrorString(result));
        return -1;
    }

    initiator_kernel<<<1, 1, 0, stream>>>(addr, size);
    result = hipGetLastError();
    if (result != hipSuccess) {
        fprintf(stderr, "[%s:%d] cuda failed with %s", __FILE__, __LINE__, hipGetErrorString(result));
        return -1;
    }

    return 0;
}

} /* extern C */
