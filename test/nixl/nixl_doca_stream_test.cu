#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <iostream>
#include <string>
#include <algorithm>
#include <nixl_descriptors.h>
#include <nixl_params.h>
#include <nixl.h>
#include <cassert>
#include "stream/metadata_stream.h"
#include "serdes/serdes.h"

#define CUDA_THREADS 512
#define TRANSFER_NUM_BUFFER 32
#define SIZE 1024
#define INITIATOR_VALUE 0xbb
#define VOLATILE(x) (*(volatile typeof(x) *)&(x))
#define INITIATOR_THRESHOLD_NS 50000 //50us
#define USE_NVTX 1

#if USE_NVTX
#include <nvtx3/nvToolsExt.h>


const uint32_t colors[] = { 0xff00ff00, 0xff0000ff, 0xffffff00, 0xffff00ff, 0xff00ffff, 0xffff0000, 0xffffffff };
const int num_colors = sizeof(colors)/sizeof(uint32_t);

#define DEVICE_GET_TIME(globaltimer) asm volatile("mov.u64 %0, %globaltimer;" : "=l"(globaltimer))

#define PUSH_RANGE(name,cid) { \
	int color_id = cid; \
	color_id = color_id%num_colors;\
	nvtxEventAttributes_t eventAttrib = {0}; \
	eventAttrib.version = NVTX_VERSION; \
	eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE; \
	eventAttrib.colorType = NVTX_COLOR_ARGB; \
	eventAttrib.color = colors[color_id]; \
	eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII; \
	eventAttrib.message.ascii = name; \
	nvtxRangePushEx(&eventAttrib); \
}
#define POP_RANGE nvtxRangePop();
#else
#define PUSH_RANGE(name,cid)
#define POP_RANGE
#endif

static void checkCudaError(hipError_t result, const char *message) {
	if (result != hipSuccess) {
		std::cerr << message << " (Error code: " << result << " - "
				   << hipGetErrorString(result) << ")" << std::endl;
		exit(EXIT_FAILURE);
	}
}

__global__ void target_kernel(uintptr_t addr, uint8_t val)
{
	uint8_t ok = 1;
	uintptr_t buffer_addr = addr + (threadIdx.x * SIZE);

	printf(">>>>>>> CUDA target waiting on buffer %d addr %lx size %d\n",
			threadIdx.x, buffer_addr, (uint32_t)SIZE);

	while(VOLATILE(((uint8_t*)buffer_addr)[0]) == 0);

	for (int i = 0; i < (int)SIZE; i++) {
		if (((uint8_t*)buffer_addr)[i] != val) {
			printf(">>>>>>> CUDA target byte %x is wrong\n", i);
			ok = 1;
		}
	}
	if (ok == 1)
		printf(">>>>>>> CUDA target, all bytes received! val=%d\n", val);
	else
		printf(">>>>>>> CUDA target, not all received bytes are ok!\n");
}

int launch_target_wait_kernel(hipStream_t stream, uintptr_t addr, uint8_t val)
{
	hipError_t result = hipSuccess;

	/* Check no previous CUDA errors */
	result = hipGetLastError();
	if (result != hipSuccess) {
		fprintf(stderr, "[%s:%d] cuda failed with %s", __FILE__, __LINE__, hipGetErrorString(result));
		return -1;
	}

	target_kernel<<<1, TRANSFER_NUM_BUFFER, 0, stream>>>(addr, val);
	result = hipGetLastError();
	if (result != hipSuccess) {
		fprintf(stderr, "[%s:%d] cuda failed with %s", __FILE__, __LINE__, hipGetErrorString(result));
		return -1;
	}

	return 0;
}

__global__ void initiator_kernel(uintptr_t addr)
{
	unsigned long long start, end;
	// Each block updates a buffer in this transfer
	uintptr_t block_address = (addr + (blockIdx.x * SIZE));

	/* Simulate a longer CUDA kernel to process initiator data */
	DEVICE_GET_TIME(start);

	for (int i = threadIdx.x; i < SIZE; i+=blockDim.x)
		((uint8_t*)block_address)[i] = INITIATOR_VALUE;

	__syncthreads();

	do {
		DEVICE_GET_TIME(end);
	} while (end - start < INITIATOR_THRESHOLD_NS);
}

int launch_initiator_send_kernel(hipStream_t stream, uintptr_t addr)
{
	hipError_t result = hipSuccess;

	/* Check no previous CUDA errors */
	result = hipGetLastError();
	if (result != hipSuccess) {
		fprintf(stderr, "[%s:%d] cuda failed with %s", __FILE__, __LINE__, hipGetErrorString(result));
		return -1;
	}

	// Block = # buffers x transfer
	initiator_kernel<<<TRANSFER_NUM_BUFFER, CUDA_THREADS, 0, stream>>>(addr);
	result = hipGetLastError();
	if (result != hipSuccess) {
		fprintf(stderr, "[%s:%d] cuda failed with %s", __FILE__, __LINE__, hipGetErrorString(result));
		return -1;
	}

	return 0;
}

/**
 * This test does p2p from using PUT.
 * intitator -> target so the metadata and
 * desc list needs to move from
 * target to initiator
 */

bool allBytesAre(void* buffer, size_t size, uint8_t value) {
	uint8_t* byte_buffer = static_cast<uint8_t*>(buffer); // Cast void* to uint8_t*
	// Iterate over each byte in the buffer
	for (size_t i = 0; i < size; ++i) {
		if (byte_buffer[i] != value) {
			return false; // Return false if any byte doesn't match the value
		}
	}
	return true; // All bytes match the value
}

std::string recvFromTarget(int port) {
	nixlMDStreamListener listener(port);
	listener.setupListenerSync();
	listener.acceptClient();
	return listener.recvFromClient();
}

void sendToInitiator(const char *ip, int port, std::string data) {
	nixlMDStreamClient client(ip, port);
	client.connectListenerSync();
	client.sendData(data);
}

int main(int argc, char *argv[]) {
	int                     peer_port;
	nixl_status_t           ret = NIXL_SUCCESS;
	uint8_t                 *data_address;
	std::string             role;
	std::string             processing;
	const char              *peer_ip;
	nixl_blob_t             remote_desc;
	nixl_blob_t             metadata;
	nixl_blob_t             remote_metadata;
	int                     status = 0;
	static std::string target("target");
	static std::string initiator("initiator");

	/** NIXL declarations */
	/** Agent and backend creation parameters */
	nixlAgentConfig cfg(true);
	nixl_b_params_t params;
	nixlBlobDesc    buf[TRANSFER_NUM_BUFFER];
	nixlBackendH    *doca;
	hipStream_t    stream;
	/** Serialization/Deserialization object to create a blob */
	nixlSerDes *serdes        = new nixlSerDes();
	nixlSerDes *remote_serdes = new nixlSerDes();

	/** Descriptors and Transfer Request */
	nixl_reg_dlist_t  dram_for_doca(DRAM_SEG);
	nixlXferReqH      *treq;
	nixl_notifs_t notifs;

	/** Argument Parsing */
	if (argc < 5) {
		std::cout <<"Enter the required arguments\n" << std::endl;
		std::cout <<"<Role> <Peer IP> <Peer Port> <CPU or GPU processing>"
				  << std::endl;
		exit(-1);
	}

	role = std::string(argv[1]);
	std::transform(role.begin(), role.end(), role.begin(), ::tolower);
	if (!role.compare(initiator) && !role.compare(target)) {
			std::cerr << "Invalid role. Use 'initiator' or 'target'."
					  << "Currently "<< role <<std::endl;
			return 1;
	}

	peer_ip   = argv[2];
	peer_port = std::stoi(argv[3]);
	processing = std::string(argv[4]);
	std::transform(processing.begin(), processing.end(), processing.begin(), ::tolower);
	if (!processing.compare("cpu") && !processing.compare("gpu")) {
			std::cerr << "Invalid type of processing. Use 'cpu' or 'gpu'."
					  << "Currently "<< processing <<std::endl;
			return 1;
	}

	/*** End - Argument Parsing */
	checkCudaError(hipSetDevice(0), "Failed to set device");
	hipFree(0);

	/** Common to both Initiator and Target */
	std::cout << "Starting Agent for "<< role << "\n";
	nixlAgent     agent(role, cfg);
	params["network_devices"] = "mlx5_0";
	params["gpu_devices"] = "0";
	PUSH_RANGE("createBackend", 0)
	agent.createBackend("DOCA", params, doca);
	POP_RANGE

	nixl_opt_args_t extra_params;
	extra_params.backends.push_back(doca);

	checkCudaError(hipMalloc(&data_address, SIZE * TRANSFER_NUM_BUFFER), "Failed to allocate CUDA buffer 0");
	checkCudaError(hipMemset((void*)data_address, 0, SIZE * TRANSFER_NUM_BUFFER), "Failed to memset CUDA buffer 0");

	if (role != target) {
		std::cout << "Allocating for initiator : "
				  << TRANSFER_NUM_BUFFER << " buffers "
				  << SIZE << " Bytes each "
				  << (void*)data_address << " address "
				  << std::endl;
	} else {
		std::cout << "Allocating for target : "
				  << TRANSFER_NUM_BUFFER << " buffers "
				  << SIZE << " Bytes each "
				  << (void*)data_address << " address "
				  << std::endl;
	}

	for (int i = 0; i < TRANSFER_NUM_BUFFER; i++) {
		buf[i].addr  = (uintptr_t)(data_address + (i * SIZE));
		buf[i].len   = SIZE;
		buf[i].devId = 0;
		dram_for_doca.addDesc(buf[i]);
	}
	/** Register memory in both initiator and target */
	agent.registerMem(dram_for_doca, &extra_params);
	agent.getLocalMD(metadata);

	std::cout << " Start Control Path metadata exchanges \n";
	if (role == target) {
		bool found = false;
		//Not used
		#if USE_FETCH_REMOTE_MD
			std::string message = serdes->exportStr();
			if (agent.genNotif(initiator, message, &extra_params) != NIXL_SUCCESS) {
				std::cout << "Can't send notif " << message << std::endl;
			}
		#else
			nixlMDStreamClient client(peer_ip, peer_port);
			client.connectListenerSync();
			nixlMDStreamListener listener(peer_port);
			listener.setupListenerSync();
		
			std::cout << " Desc List from Target to Initiator\n";
			dram_for_doca.print();

			//Send local MD to remote initiator
			assert(serdes->addStr("AgentMD", metadata) == NIXL_SUCCESS);
			assert(dram_for_doca.trim().serialize(serdes) == NIXL_SUCCESS);
			client.sendData(serdes->exportStr());
			std::cout << " End Control Path metadata exchanges \n";

			//Wait to receive remote MD from remote initiator
			//Not required by DOCA Backend but needed by Agent to populate remoteBackends array
			//Without this step, can't call genNotif() from target as the remoteBackends doesn't
			//have populated initiator entry.
			listener.acceptClient();
			std::string rrstr = listener.recvFromClient();
			remote_serdes->importStr(rrstr);
			remote_metadata = remote_serdes->getStr("AgentMD");
			assert (remote_metadata != "");
			agent.loadRemoteMD(remote_metadata, initiator);

			std::cout << " Serialize Metadata to string and Send to Initiator\n";
			std::cout << " \t -- To be handled by runtime - currently sent via a TCP Stream\n";
		#endif

		//First recv notif: initiator ack it connected correctly
		do {
			nixl_status_t ret = agent.getNotifs(notifs);
		} while(notifs.size() == 0);

		for (const auto& n : notifs) {
			for (size_t idx = 0; idx < n.second.size(); idx++) {
				std::cout << "Received message from " << n.first << " msg: " << n.second[idx] << " at " << idx << std::endl;

				if (n.first == initiator && n.second[idx] == "connected") {
					std::cout << "Received correct message from " << n.first << " msg: " << n.second[idx] << " at " << idx << std::endl;
					break;
				}
			}
		}

		std::cout << " Start Data Path Exchanges \n";
		std::cout << " Waiting to receive Data from Initiator\n";

		checkCudaError(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking), "Failed to create CUDA stream");
	
		//Second recv notif: initiator ack data has been sent
		do {
			for (const auto& n : notifs) {
				for (size_t idx = 0; idx < n.second.size(); idx++) {
					if (n.first == initiator && n.second[idx] == "sent") {
						std::cout << "Received correct message from " << n.first << " msg: " << n.second[idx] << " at " << idx << std::endl;
						launch_target_wait_kernel(stream, (uintptr_t)(data_address), INITIATOR_VALUE);
						hipStreamSynchronize(stream);
						std::cout << " DOCA Transfer completed -- first!\n";
						found = true;
						break;
					}
				}
			}
			nixl_status_t ret = agent.getNotifs(notifs);
		} while (found == false);

		notifs.clear();

		//First send notif: target processed previously sent data
		std::string msg = "processed";
		ret = agent.genNotif(initiator, msg, &extra_params);
		if(ret != NIXL_SUCCESS) {
			std::cerr << "Target genNotif error " << ret << "\n";
		}
		found = false;

		std::cout << " Waiting for second 'sent' notif\n";
		//Third recv notif: sent
		do {
			for (const auto& n : notifs) {
				for (size_t idx = 0; idx < n.second.size(); idx++) {
					if (n.first == initiator && n.second[idx] == "sent") {
						std::cout << "Received correct message from " << n.first << " msg: " << n.second[idx] << " at " << idx << std::endl;
						launch_target_wait_kernel(stream, (uintptr_t)(data_address), INITIATOR_VALUE+1);
						hipStreamSynchronize(stream);
						std::cout << " DOCA Transfer completed -- second!\n";
						found = true;
						break;
					}
				}
			}
			nixl_status_t ret = agent.getNotifs(notifs);
			} while (found == false);

		hipStreamDestroy(stream);
	} else {
		std::cout << " Wait for metadata from Target \n";
		std::cout << " \t -- To be handled by runtime - currently received via a TCP Stream\n";
		
		//Not used
		#if USE_FETCH_REMOTE_MD
			nixl_opt_args_t md_extra_params;
			md_extra_params.ipAddr = peer_ip;
			md_extra_params.port = peer_port;
			agent.fetchRemoteMD(target, &md_extra_params);
			agent.sendLocalMD(&md_extra_params);

			do {
				nixl_status_t ret = agent.getNotifs(notifs);
			} while(notifs.size() == 0);

			for (const auto &notif : notifs[target]) {
				remote_serdes->importStr(notif);
			}

			for (const auto& n : notifs) {
				if (n.first == target && n.second[0] == "connected") {
					std::cout << "Received correct message from " << n.first << " msg: " << n.second[0] << std::endl;
					break;
				} else {
					std::cout << "Received wrong message from " << n.first << " msg: " << n.second[0] << std::endl;
				}
			}
		#else
			//Wait for remote target connection
			nixlMDStreamListener listener(peer_port);
			listener.setupListenerSync();
			listener.acceptClient();
			std::string rrstr = listener.recvFromClient();
			remote_serdes->importStr(rrstr);
			remote_metadata = remote_serdes->getStr("AgentMD");
			assert (remote_metadata != "");
			agent.loadRemoteMD(remote_metadata, target);

			//Wait target to open listener
			sleep(2);
			//Send to remote target local connection info.
			//Not needed by DOCA backend, required by NIXL Agent (see above)
			nixlMDStreamClient client(peer_ip, peer_port);
			client.connectListenerSync();
			assert(serdes->addStr("AgentMD", metadata) == NIXL_SUCCESS);
			assert(dram_for_doca.trim().serialize(serdes) == NIXL_SUCCESS);
			client.sendData(serdes->exportStr());
			std::cout << " End Control Path metadata exchanges \n";
		#endif

		//First send notif: connected
		std::string msg = "connected";
		ret = agent.genNotif(target, msg);
		if(ret != NIXL_SUCCESS) {
			std::cerr << "Target genNotif error " << ret << "\n";
		}

		std::cout << " Verify Deserialized Target's Desc List at Initiator\n";
		nixl_xfer_dlist_t dram_target_doca(remote_serdes);
		nixl_xfer_dlist_t dram_initiator_doca = dram_for_doca.trim();
		dram_target_doca.print();
		std::cout << " Got metadata from " << target << " \n";
		std::cout << " Create transfer request with DOCA backend\n ";

		PUSH_RANGE("createXferReq", 1)

		//Create Xfer request with notification
		if (processing.compare("gpu") == 0)
	        checkCudaError(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking), "Failed to create CUDA stream");

		if (processing.compare("gpu") == 0) {
			extra_params.customParam.resize(sizeof(uintptr_t));
			*((uintptr_t*) extra_params.customParam.data()) = (uintptr_t)stream;
		}
		extra_params.notifMsg = "sent";
		extra_params.hasNotif = true;
		ret = agent.createXferReq(NIXL_WRITE, dram_initiator_doca, dram_target_doca,
						target, treq, &extra_params);
		if (ret != NIXL_SUCCESS) {
			std::cerr << "Error creating transfer request\n";
			exit(-1);
		}
		POP_RANGE

		std::cout << "Launch initiator send kernel on stream\n";

		/* Synthetic simulation of GPU processing data before sending */
		if (processing.compare("gpu") == 0) {
			std::cout << " Prepare data, GPU mode, transfer 1" << std::endl;
			PUSH_RANGE("InitData", 2)
			launch_initiator_send_kernel(stream, (uintptr_t)(data_address));
			POP_RANGE

			std::cout << " Post the request with DOCA backend transfer 1" << std::endl;
			PUSH_RANGE("postXferReq", 3)
			status = agent.postXferReq(treq);
			assert(status >= NIXL_SUCCESS);
			POP_RANGE
		} else {
			/* Synthetic simulation of CPU processing data before sending */
			std::cout << "First xfer, prepare data, CPU mode, transfer 1" << std::endl;
			PUSH_RANGE("InitData", 2)
			hipMemset((void*)data_address, INITIATOR_VALUE, TRANSFER_NUM_BUFFER * SIZE);
			POP_RANGE

			std::cout << " Post the request with DOCA backend transfer 1" << std::endl;
			PUSH_RANGE("postXferReq", 3)
			status = agent.postXferReq(treq);
			assert(status >= NIXL_SUCCESS);
			POP_RANGE

			std::cout << " Waiting for completion\n";
			PUSH_RANGE("getXferStatus", 4)
			while (status != NIXL_SUCCESS) {
				status = agent.getXferStatus(treq);
				assert(status >= NIXL_SUCCESS);
			}
			POP_RANGE

			std::cout << "Second xfer, prepare data, CPU mode, transfer 2" << std::endl;
			PUSH_RANGE("InitData", 2)
			hipMemset((void*)data_address, INITIATOR_VALUE + 1, TRANSFER_NUM_BUFFER * SIZE);
			POP_RANGE

			//First recv notif: target processed previously sent data
			do {
				nixl_status_t ret = agent.getNotifs(notifs);
			} while(notifs.size() == 0);

			for (const auto& n : notifs) {
				for (size_t idx = 0; idx < n.second.size(); idx++) {
					if (n.first == target && n.second[idx] == "processed") {
						std::cout << "Received correct message from " << n.first << " msg: " << n.second[idx] << " at " << idx << std::endl;
						break;
					}
				}
			}

			//Repost same treq with different data in buffers
			std::cout << " Post the request with DOCA backend transfer 2" << std::endl;
			PUSH_RANGE("postXferReq", 3)
			status = agent.postXferReq(treq);
			assert(status >= NIXL_SUCCESS);
			POP_RANGE

			std::cout << " Waiting for completion\n";
			PUSH_RANGE("getXferStatus", 4)
			while (status != NIXL_SUCCESS) {
				status = agent.getXferStatus(treq);
				assert(status >= NIXL_SUCCESS);
			}
			POP_RANGE
		}

		std::cout << "Releasing request " << std::endl;
		agent.releaseXferReq(treq);
	
		if (processing.compare("gpu") == 0) {
			hipStreamSynchronize(stream);
			hipStreamDestroy(stream);
		}
	}

	std::cout <<"Cleanup.. \n";
	
	agent.deregisterMem(dram_for_doca, &extra_params);
	// hipFree(data_address);

	if (role == "target")
		delete serdes;
	else
		delete remote_serdes;

	std::cout <<"Exit.. \n";

	return 0;
}
